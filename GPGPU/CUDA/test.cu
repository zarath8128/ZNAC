#include <iostream>
#include <hip/hip_runtime.h>

int main()
{
	hipError_t err;
	double *v;
	int c;
	hipGetDeviceCount(&c);
	std::cout << c << std::endl;
	err = hipMalloc(&v, 100*sizeof(double));
	std::cout << hipGetErrorString(err) << std::endl;
	err = hipFree(v);
	std::cout << hipGetErrorString(err) << std::endl;
}
